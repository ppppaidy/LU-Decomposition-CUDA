#include <iostream>
#include <fstream>
#include <sstream>
#include <malloc.h>
#include <time.h>
#include <algorithm>
#include <string>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <vector>
#include <hip/hip_runtime.h>

char input_buf[256];
std::string L_outfile, U_outfile, infile; 
int N, Num;
int *x_pos, *y_pos;
double *val;

__global__ void LU_Decompose(int *head, double *A, double *L, int *x_pos, int N, int Num){
    int total = gridDim.x * blockDim.x;
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    for(int i = 0; i < N; i++){
        int h_start = head[i];
        int h_end = head[i+1];
        double uii = A[h_start];
        if(id == 0) L[h_start] = 1;
        for(int j = h_start + 1 + id; j < h_end; j += total){
            double lj = A[j]/uii;
            L[j] = lj;
            int pos = x_pos[j];
            int p = head[pos];
            for(int k = j; k < h_end; k++){
                int qos = x_pos[k];
                while(x_pos[p] != qos) p++;
                A[p] -= lj * A[k];
            }
        }
        __syncthreads();
    }
}

int main(int argc, char* argv[]){
    // handle args
    if(argc != 2){
        std::cerr<<"usage : "<<argv[0]<<" [matrix_file]"<<std::endl;
        return -1;
    }
    std::ifstream mtx_in(argv[1]);
    if(!mtx_in.is_open()){
        std::cerr<<"can not open "<<argv[1]<<std::endl;
        return -1;
    }
    infile = argv[1];
    if(infile.find_last_of("/") != infile.npos){
        infile = infile.substr(infile.find_last_of("/") + 1);
    }
    if(access("result", 4) != 0){
        mkdir("result", 0777);
    }
    L_outfile = "result/L3_" + infile;
    U_outfile = "result/U3_" + infile;

    // input matrix
    while(1){
        mtx_in.getline(input_buf, 256, '\n');
        if(input_buf[0] == '%') continue;
        std::stringstream sin;
        sin<<input_buf;
        sin>>N>>N>>Num;
        break;
    }
    x_pos = (int*)malloc(Num * sizeof(int));
    y_pos = (int*)malloc(Num * sizeof(int));
    val = (double*)malloc(Num * sizeof(double));
    for(int i = 0; i < Num; i++){
        mtx_in>>x_pos[i]>>y_pos[i]>>val[i];
        x_pos[i]--;
        y_pos[i]--;
    }
    mtx_in.close();

    // LU Decompose start

    clock_t start = clock();

    int *head = (int*)malloc((N+1) * sizeof(int));
    int A_Num = 0;

    // fill in
    std::vector<std::vector<int> > fill_in(N);
    for(int i = 0; i < Num; i++){
        fill_in[y_pos[i]].push_back(x_pos[i]);
        if(i == Num || y_pos[i] != y_pos[i+1]){
            int pos = y_pos[i];
            std::sort(fill_in[pos].begin(), fill_in[pos].end());
            std::vector<int>::iterator tail = std::unique(fill_in[pos].begin(), fill_in[pos].end());
            fill_in[pos].erase(tail, fill_in[pos].end());
            if(fill_in[pos].size() >= 3){
                int fat = fill_in[pos][1];
                for(int j = 2; j < fill_in[pos].size(); j++){
                    fill_in[fat].push_back(fill_in[pos][j]);
                }
            }
            head[pos] = A_Num;
            A_Num += fill_in[pos].size();
        }
    }
    head[N] = A_Num;

    clock_t middle = clock();
    std::cerr<<"Task C of "<<argv[1]<<" costs "<<(double)(middle-start)/CLOCKS_PER_SEC<<"s to fill in."<<std::endl;

    double* A = (double*)malloc(A_Num * sizeof(double));
    double* L = (double*)malloc(A_Num * sizeof(double));

    int *x_pos_new = (int*)malloc(A_Num * sizeof(int));

    A_Num = 0; Num = 0;
    for(int i = 0; i < N; i++){
        for(int j = 0; j < fill_in[i].size(); j++){
            x_pos_new[A_Num] = fill_in[i][j];
            if(fill_in[i][j] == x_pos[Num] && i == y_pos[Num]){
                A[A_Num] = val[Num++];
            }
            else{
                A[A_Num] = 0;
            }
            A_Num ++;
        }
    }

    int* head_cuda;
    double* A_cuda, *L_cuda;
    int* x_pos_new_cuda;
    hipMalloc((void**)&head_cuda, (N+1) * sizeof(int));
    hipMalloc((void**)&A_cuda, A_Num * sizeof(double));
    hipMalloc((void**)&L_cuda, A_Num * sizeof(double));
    hipMalloc((void**)&x_pos_new_cuda, A_Num * sizeof(int));

    hipMemcpy(head_cuda, head, (N+1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(A_cuda, A, A_Num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(x_pos_new_cuda, x_pos_new, A_Num * sizeof(int), hipMemcpyHostToDevice);

    int block_num = 1;
    int thread_num = 1024;

    LU_Decompose<<<block_num, thread_num>>>(head_cuda, A_cuda, L_cuda, x_pos_new_cuda, N, A_Num);

    hipMemcpy(A, A_cuda, A_Num * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(L, L_cuda, A_Num * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(head_cuda);
    hipFree(A_cuda);
    hipFree(L_cuda);
    hipFree(x_pos_new_cuda);
    
    clock_t end = clock();

    // LU Decompose end

    std::cerr<<"Task C of "<<argv[1]<<" costs "<<(double)(end-start)/CLOCKS_PER_SEC<<"s."<<std::endl;

    // output L & U
    if(A_Num < 3000000){
        std::ofstream L_out(L_outfile.c_str()), U_out(U_outfile.c_str());
        L_out<<N<<" "<<N<<" "<<A_Num<<std::endl;
        for(int i = 0; i < N; i++){
            for(int j = head[i]; j < head[i+1]; j++){
                L_out<<x_pos_new[j]+1<<" "<<i+1<<" "<<L[j]<<std::endl;
            }
        }
        U_out<<N<<" "<<N<<" "<<A_Num<<std::endl;
        for(int i = 0; i < N; i++){
            for(int j = head[i]; j < head[i+1]; j++){
                U_out<<i+1<<" "<<x_pos_new[j]+1<<" "<<A[j]<<std::endl;
            }
        }
        L_out.close();
        U_out.close();
    }

    free(head);
    free(A);
    free(L);
    free(x_pos_new);

    free(x_pos);
    free(y_pos);
    free(val);

    return 0;
}