
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <malloc.h>
#include <time.h>
#include <algorithm>
#include <string>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>

char input_buf[256];
std::string L_outfile, U_outfile, infile; 
int N, Num;
int *x_pos, *y_pos;
double *val, *A, *L, *U;
const double eps = 0;

void LU_Decompose(double *A, double *L, double *U, int N){
    for(int i = 0; i < N; i++){
        L[i*N + i] = 1;
        for(int j = i; j < N; j++){
            double SU = 0;
            for(int k = 0; k < i; k++){
                SU += L[i*N + k] * U[k*N + j];
            }
            U[i*N + j] = A[i*N + j] - SU;
            if(j + 1 < N){
                double SL = 0;
                for(int k = 0; k < i; k++){
                    SL += L[(j+1)*N + k] * U[k*N + i];
                }
                L[(j+1)*N + i] = (A[(j+1)*N + i] - SL)/U[i*N + i];
            }
        }
    }
}

int main(int argc, char* argv[]){
    // handle args
    if(argc != 2){
        std::cerr<<"usage : "<<argv[0]<<" [matrix_file]"<<std::endl;
        return -1;
    }
    std::ifstream mtx_in(argv[1]);
    if(!mtx_in.is_open()){
        std::cerr<<"can not open "<<argv[1]<<std::endl;
        return -1;
    }
    infile = argv[1];
    if(infile.find_last_of("/") != infile.npos){
        infile = infile.substr(infile.find_last_of("/") + 1);
    }
    if(access("result", 4) != 0){
        mkdir("result", 0777);
    }
    L_outfile = "result/L1_" + infile;
    U_outfile = "result/U1_" + infile;

    // input matrix
    while(1){
        mtx_in.getline(input_buf, 256, '\n');
        if(input_buf[0] == '%') continue;
        std::stringstream sin;
        sin<<input_buf;
        sin>>N>>N>>Num;
        break;
    }
    x_pos = (int*)malloc(Num * sizeof(int));
    y_pos = (int*)malloc(Num * sizeof(int));
    val = (double*)malloc(Num * sizeof(double));
    for(int i = 0; i < Num; i++){
        mtx_in>>x_pos[i]>>y_pos[i]>>val[i];
        x_pos[i]--;
        y_pos[i]--;
    }
    mtx_in.close();

    A = (double*)malloc(N * N * sizeof(double));
    L = (double*)malloc(N * N * sizeof(double));
    U = (double*)malloc(N * N * sizeof(double));

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            A[i*N + j] = 0;
            L[i*N + j] = 0;
            U[i*N + j] = 0;
        }
    }
    for(int i = 0; i < Num; i++){
        A[x_pos[i]*N + y_pos[i]] = val[i];
        A[y_pos[i]*N + x_pos[i]] = val[i];
    }

    // LU Decompose start

    clock_t start = clock();

    LU_Decompose(A, L, U, N);
    
    clock_t end = clock();

    // LU Decompose end

    std::cerr<<"Task A of "<<argv[1]<<" costs "<<(double)(end-start)/CLOCKS_PER_SEC<<"s."<<std::endl;

    // output L & U
    std::ofstream L_out(L_outfile.c_str()), U_out(U_outfile.c_str());
    int L_Num = 0, U_Num = 0;
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            if(abs(L[i*N + j]) > eps) L_Num ++;
        }
    }
    L_out<<N<<" "<<N<<" "<<L_Num<<std::endl;
    for(int j = 0; j < N; j++){
        for(int i = 0; i < N; i++){
            if(abs(L[i*N + j]) > eps){
                L_out<<i+1<<" "<<j+1<<" "<<L[i*N + j]<<std::endl;
            }
        }
    }
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            if(abs(U[i*N + j]) > eps) U_Num ++;
        }
    }
    U_out<<N<<" "<<N<<" "<<U_Num<<std::endl;
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            if(abs(U[i*N + j]) > eps){
                U_out<<i+1<<" "<<j+1<<" "<<U[i*N + j]<<std::endl;
            }
        }
    }
    L_out.close();
    U_out.close();

    free(A);
    free(L);
    free(U);

    free(x_pos);
    free(y_pos);
    free(val);

    return 0;
}