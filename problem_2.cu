
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <malloc.h>
#include <time.h>
#include <algorithm>
#include <string>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <vector>
#include <unordered_map>

char input_buf[256];
std::string L_outfile, U_outfile, infile; 
int N, Num;
int *x_pos, *y_pos;
double *val;

int main(int argc, char* argv[]){
    // handle args
    if(argc != 2){
        std::cerr<<"usage : "<<argv[0]<<" [matrix_file]"<<std::endl;
        return -1;
    }
    std::ifstream mtx_in(argv[1]);
    if(!mtx_in.is_open()){
        std::cerr<<"can not open "<<argv[1]<<std::endl;
        return -1;
    }
    infile = argv[1];
    if(infile.find_last_of("/") != infile.npos){
        infile = infile.substr(infile.find_last_of("/") + 1);
    }
    if(access("result", 4) != 0){
        mkdir("result", 0777);
    }
    L_outfile = "result/L2_" + infile;
    U_outfile = "result/U2_" + infile;

    // input matrix
    while(1){
        mtx_in.getline(input_buf, 256, '\n');
        if(input_buf[0] == '%') continue;
        std::stringstream sin;
        sin<<input_buf;
        sin>>N>>N>>Num;
        break;
    }
    x_pos = (int*)malloc(Num * sizeof(int));
    y_pos = (int*)malloc(Num * sizeof(int));
    val = (double*)malloc(Num * sizeof(double));
    for(int i = 0; i < Num; i++){
        mtx_in>>x_pos[i]>>y_pos[i]>>val[i];
        x_pos[i]--;
        y_pos[i]--;
    }
    mtx_in.close();

    // LU Decompose start

    clock_t start = clock();

    std::vector<std::unordered_map<int, double> > L_val(N);
    std::vector<std::unordered_map<int, double> > U_val(N);
    for(int i = 0; i < Num; i++){
        U_val[y_pos[i]][x_pos[i]] += val[i];
        L_val[y_pos[i]][x_pos[i]] += val[i];
        if(i == Num-1 || y_pos[i] != y_pos[i+1]){
            double uii = U_val[y_pos[i]][y_pos[i]];
            for(std::unordered_map<int, double>::iterator it = L_val[y_pos[i]].begin(); it != L_val[y_pos[i]].end(); it++){
                it->second /= uii;
            }
            L_val[y_pos[i]][y_pos[i]] = 1;
        }
        if(i < Num-1 && y_pos[i] != y_pos[i+1]){
            for(std::unordered_map<int, double>::iterator it1 = L_val[y_pos[i]].begin(); it1 != L_val[y_pos[i]].end(); it1++){
                for(std::unordered_map<int, double>::iterator it2 = U_val[y_pos[i]].begin(); it2 != U_val[y_pos[i]].end(); it2++){
                    if(it1->first != y_pos[i] && it2->first != y_pos[i]){
                        if(it1->first <= it2->first){
                            U_val[it1->first][it2->first] -= it1->second * it2->second;
                        }
                        else{
                            L_val[it2->first][it1->first] -= it1->second * it2->second;
                        }
                    }
                }
            }
        }
    }
    
    clock_t end = clock();

    // LU Decompose end

    std::cerr<<"Task B of "<<argv[1]<<" costs "<<(double)(end-start)/CLOCKS_PER_SEC<<"s."<<std::endl;

    // output L & U
    std::ofstream L_out(L_outfile.c_str()), U_out(U_outfile.c_str());
    int L_Num = 0, U_Num = 0;
    for(int i = 0; i < N; i++){
        for(std::unordered_map<int, double>::iterator it = U_val[i].begin(); it != U_val[i].end(); it++){
            U_Num ++;
        }
    }
    L_Num = U_Num;
    U_out<<N<<" "<<N<<" "<<U_Num<<std::endl;
    for(int i = 0; i < N; i++){
        for(std::unordered_map<int, double>::iterator it = U_val[i].begin(); it != U_val[i].end(); it++){
            U_out<<i+1<<" "<<it->first+1<<" "<<it->second<<std::endl;
        }
    }
    L_out<<N<<" "<<N<<" "<<L_Num<<std::endl;
    for(int i = 0; i < N; i++){
        for(std::unordered_map<int, double>::iterator it = L_val[i].begin(); it != L_val[i].end(); it++){
            L_out<<it->first+1<<" "<<i+1<<" "<<it->second<<std::endl;
        }
    }
    L_out.close();
    U_out.close();

    free(x_pos);
    free(y_pos);
    free(val);

    return 0;
}